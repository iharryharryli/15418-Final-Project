#include "hip/hip_runtime.h"
#include "depend.h"

struct particles_t
{
  double *x;
  double *y;
  double *z;

  int spawn_rate;
  int curr_particles;
  int max_particles;
  int num_particles;
};

__constant__ particles_t particles;
particles_t particles_cpu;

__device__ __inline__ void 
StaggeredVelocity(double px, double py, double pz,
    double* ux, double* uy, double* uz)
// evaluates velocity at (px,py,pz) in the grid torus with staggered
// velocity vector field vx,vy,vz
{
    px = fmod(px, (double)torus.sizex);
    py = fmod(py, (double)torus.sizey);
    pz = fmod(pz, (double)torus.sizez);

    
    /*if (px < 0 || px > torus.sizex 
     || py < 0 || py > torus.sizey
     || pz < 0 || pz > torus.sizez)
    {
        *ux = 0;
        *uy = 0;
        *uz = 0;
        return;
    }*/

    // ix = floor(px/torus.dx) + 1;
    // iy = floor(py/torus.dy) + 1;
    // iz = floor(pz/torus.dz) + 1;

    int ix = floor(px / torus.dx);
    int iy = floor(py / torus.dy);
    int iz = floor(pz / torus.dz);

    // ixp = mod(ix,torus.resx)+1;
    // iyp = mod(iy,torus.resy)+1;
    // izp = mod(iz,torus.resz)+1;

    int ixp = (ix + 1) % torus.resx;
    int iyp = (iy + 1) % torus.resy;
    int izp = (iz + 1) % torus.resz;

    // ind0 = sub2ind([torus.resx,torus.resy,torus.resz],ix,iy,iz);
    // indxp = sub2ind([torus.resx,torus.resy,torus.resz],ixp,iy,iz);
    // indyp = sub2ind([torus.resx,torus.resy,torus.resz],ix,iyp,iz);
    // indzp = sub2ind([torus.resx,torus.resy,torus.resz],ix,iy,izp);
    // indxpyp = sub2ind([torus.resx,torus.resy,torus.resz],ixp,iyp,iz);
    // indypzp = sub2ind([torus.resx,torus.resy,torus.resz],ix,iyp,izp);
    // indxpzp = sub2ind([torus.resx,torus.resy,torus.resz],ixp,iy,izp);

    int ind0 =    index3d(ix, iy, iz);
    int indxp =   index3d(ixp, iy, iz);
    int indyp =   index3d(ix, iyp, iz);
    int indzp =   index3d(ix, iy, izp); 
    int indxpyp = index3d(ixp, iyp, iz);
    int indypzp = index3d(ix, iyp, izp);
    int indxpzp = index3d(ixp, iy, izp);    
    
    // wx = px - (ix-1)*torus.dx;
    // wy = py - (iy-1)*torus.dy;
    // wz = pz - (iz-1)*torus.dz;

    double wx = px - ix * torus.dx;
    double wy = py - iy * torus.dy;
    double wz = pz - iz * torus.dz;

    // ux = (1-wz).*((1-wy).*vx(ind0 )+wy.*vx(indyp  )) + ...
    //         wz .*((1-wy).*vx(indzp)+wy.*vx(indypzp));
    // uy = (1-wz).*((1-wx).*vy(ind0 )+wx.*vy(indxp  )) + ...
    //         wz .*((1-wx).*vy(indzp)+wx.*vy(indxpzp));
    // uz = (1-wy).*((1-wx).*vz(ind0 )+wx.*vz(indxp  )) + ...
    //         wy .*((1-wx).*vz(indyp)+wx.*vz(indxpyp));

    // Interpolate between velocities at grid points
    *ux = (1 - wz) * ((1 - wy) * torus.vx[ind0] + wy * torus.vx[indyp]) 
              + wz * ((1 - wy) * torus.vx[indzp] + wy * torus.vx[indypzp]);
    *uy = (1 - wx) * ((1 - wz) * torus.vy[ind0] + wz * torus.vy[indzp]) 
              + wx * ((1 - wz) * torus.vy[indxp] + wz * torus.vy[indxpzp]);
    *uz = (1 - wy) * ((1 - wx) * torus.vz[ind0] + wx * torus.vz[indxp]) 
              + wy * ((1 - wx) * torus.vz[indyp] + wx * torus.vz[indxpyp]);
}

__global__ void StaggeredAdvect_kernel()
// advect particle positions using RK4 in a grid torus with
// staggered velocity vx,vy,vz, for dt period of time
{
    int i = check_limit(particles.num_particles);
    if(i<0)return;
    double k1x, k1y, k1z;
    double k2x, k2y, k2z;
    double k3x, k3y, k3z;
    double k4x, k4y, k4z;
    double *x = particles.x;
    double *y = particles.y;
    double *z = particles.z;
    double dt = isf.dt;
    //for (int i=0; i<particles.num_particles; i++)
    //{
        // Fourth-order Runge-Kutta method
        StaggeredVelocity(x[i], y[i], z[i], &k1x, &k1y, &k1z);
        //x[i] = k1x; y[i] = k1y; z[i] = k1z; continue;
        StaggeredVelocity(x[i]+dt*k1x/2, y[i]+dt*k1y/2, z[i]+dt*k1z/2, 
            &k2x, &k2y, &k2z);
        StaggeredVelocity(x[i]+dt*k2x/2, y[i]+dt*k2y/2, z[i]+dt*k2z/2,
            &k3x, &k3y, &k3z);
        StaggeredVelocity(x[i]+dt*k3x, y[i]+dt*k3y, z[i]+dt*k3z, 
            &k4x, &k4y, &k4z);
        x[i] += dt/6*(k1x+2*k2x+2*k3x+k4x);
        y[i] += dt/6*(k1y+2*k2y+2*k3y+k4y);
        z[i] += dt/6*(k1z+2*k2z+2*k3z+k4z);
    //}
}

void StaggeredAdvect()
{
  tpstart(10);
  int nb = calc_numblock(particles_cpu.num_particles, THREADS_PER_BLOCK);
  StaggeredAdvect_kernel<<<nb,THREADS_PER_BLOCK>>>();
  hipDeviceSynchronize(); 
  tpend(10); 
}
