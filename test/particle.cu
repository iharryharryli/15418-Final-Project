#include "hip/hip_runtime.h"
#include "depend.h"
#include "mycomplex.cu"

struct particles_t
{
  double *x;
  double *y;
  double *z;
};

__constant__ particles_t particles;
particles_t particles_cpu;

__global__ void StaggeredAdvect()
// advect particle positions using RK4 in a grid torus with
// staggered velocity vx,vy,vz, for dt period of time
{
    // [k1x,k1y,k1z] =...
    //     particle.StaggeredVelocity(...
    //         particle.x,particle.y,particle.z,...
    //         torus,vx,vy,vz);
    // [k2x,k2y,k2z] =...
    //     particle.StaggeredVelocity(...
    //         particle.x+k1x*dt/2,particle.y+k1y*dt/2,particle.z+k1z*dt/2,...
    //         torus,vx,vy,vz);
    // [k3x,k3y,k3z] =...
    //     particle.StaggeredVelocity(...
    //         particle.x+k2x*dt/2,particle.y+k2y*dt/2,particle.z+k2z*dt/2,...
    //         torus,vx,vy,vz);
    // [k4x,k4y,k4z] =...
    //     particle.StaggeredVelocity(...
    //         particle.x+k3x*dt,particle.y+k3y*dt,particle.z+k3z*dt,...
    //         torus,vx,vy,vz);
    // particle.x = particle.x + dt/6*(k1x+2*k2x+2*k3x+k4x);
    // particle.y = particle.y + dt/6*(k1y+2*k2y+2*k3y+k4y);
    // particle.z = particle.z + dt/6*(k1z+2*k2z+2*k3z+k4z);
}

__global__ void StaggeredVelocity()
// evaluates velocity at (px,py,pz) in the grid torus with staggered
// velocity vector field vx,vy,vz
{
    // px = mod(px,torus.sizex);
    // py = mod(py,torus.sizey);
    // pz = mod(pz,torus.sizez);
    
    // ix = floor(px/torus.dx) + 1;
    // iy = floor(py/torus.dy) + 1;
    // iz = floor(pz/torus.dz) + 1;
    // ixp = mod(ix,torus.resx)+1;
    // iyp = mod(iy,torus.resy)+1;
    // izp = mod(iz,torus.resz)+1;
    // ind0 = sub2ind([torus.resx,torus.resy,torus.resz],ix,iy,iz);
    // indxp = sub2ind([torus.resx,torus.resy,torus.resz],ixp,iy,iz);
    // indyp = sub2ind([torus.resx,torus.resy,torus.resz],ix,iyp,iz);
    // indzp = sub2ind([torus.resx,torus.resy,torus.resz],ix,iy,izp);
    // indxpyp = sub2ind([torus.resx,torus.resy,torus.resz],ixp,iyp,iz);
    // indypzp = sub2ind([torus.resx,torus.resy,torus.resz],ix,iyp,izp);
    // indxpzp = sub2ind([torus.resx,torus.resy,torus.resz],ixp,iy,izp);
    
    // wx = px - (ix-1)*torus.dx;
    // wy = py - (iy-1)*torus.dy;
    // wz = pz - (iz-1)*torus.dz;
    // ux = (1-wz).*((1-wy).*vx(ind0 )+wy.*vx(indyp  )) + ...
    //         wz .*((1-wy).*vx(indzp)+wy.*vx(indypzp));
    // uy = (1-wz).*((1-wx).*vy(ind0 )+wx.*vy(indxp  )) + ...
    //         wz .*((1-wx).*vy(indzp)+wx.*vy(indxpzp));
    // uz = (1-wy).*((1-wx).*vz(ind0 )+wx.*vz(indxp  )) + ...
    //         wy .*((1-wx).*vz(indyp)+wx.*vz(indxpyp));

}