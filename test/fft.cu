#include "hip/hip_runtime.h"
#include "depend.h"

__global__ void printfft(hipfftComplex *data, int len)
{
  for(int i=0; i<len; i++)
  {
    printf("%f %f \n", data[i].x, data[i].y);
  }
}

void fft()
{
  int len = 10;
  hipfftComplex A[len];
  for(int i=0; i<len; i++)
    A[i] = make_hipFloatComplex(i,0.0);
  
  hipfftComplex* cudamem;
  hipMalloc(&cudamem, sizeof(hipfftComplex) * len);
  hipMemcpy(cudamem, A, sizeof(hipfftComplex)*len, hipMemcpyHostToDevice);

  hipfftComplex *data;
  hipMalloc(&data, sizeof(hipfftComplex) * len);

  hipfftHandle plan;
  hipfftPlan1d(&plan, len, HIPFFT_C2C, 1);

  hipfftExecC2C(plan, cudamem, data, HIPFFT_FORWARD);

  hipDeviceSynchronize();

  printfft<<<1,1>>>(data,len);

  hipDeviceSynchronize();


  printf("done\n");

  
}
