#include "hip/hip_runtime.h"
#include "depend.h"

__global__ void printfft(hipfftDoubleComplex *data, int len)
{
  for(int i=0; i<len; i++)
  {
    printf("%f %f \n", data[i].x, data[i].y);
  }
}

void fft()
{
  int len = 10;
  hipfftDoubleComplex A[len];
  for(int i=0; i<len; i++)
    A[i] = make_hipDoubleComplex(i,0.0);
  
  hipfftDoubleComplex* cudamem;
  hipMalloc(&cudamem, sizeof(hipfftDoubleComplex) * len);
  hipMemcpy(cudamem, A, sizeof(hipfftDoubleComplex)*len, 
      hipMemcpyHostToDevice);

  hipfftDoubleComplex *data;
  hipMalloc(&data, sizeof(hipfftDoubleComplex) * len);

  hipfftHandle plan;
  hipfftPlan2d(&plan, 2, 5, HIPFFT_Z2Z);

  hipfftExecZ2Z(plan, cudamem, data, HIPFFT_FORWARD);

  hipDeviceSynchronize();

  hipfftExecZ2Z(plan, data, cudamem, HIPFFT_BACKWARD); 

  hipDeviceSynchronize();

  printfft<<<1,1>>>(cudamem,len);

  hipDeviceSynchronize();


  printf("done\n");

  
}
