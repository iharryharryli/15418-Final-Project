#include "hip/hip_runtime.h"
#include "ISF.cu"


struct nozzle_t
{
  float center[3];
  float len;
  float rad;
};



struct para_t
{
  float jet_velocity[3];
  nozzle_t nozzle;

  char* isJet;

  mycomplex* psi1;
  mycomplex* psi2;

};

__constant__ para_t para;

__global__ void set_nozzle()
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);
        float px = i * torus.dx;
        float py = j * torus.dy;
        float pz = k * torus.dz;

        float dx = px - para.nozzle.center[0];
        float dy = py - para.nozzle.center[1];
        float dz = pz - para.nozzle.center[2];

        if(abs(dx) < para.nozzle.len / 2.0
            && (dy * dy + dz * dz) < para.nozzle.rad * para.nozzle.rad)
        {
          para.isJet[ind] = 1;
          printf("%d %d %d \n", i, j ,k);
        }
        else
        {
          para.isJet[ind] = 0;
        }
      }
    }
  }

}

__global__ void psi_init_cuda()
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);
      }
    }
  }
}


void para_init(Torus* p, para_t* t)
{
  hipMalloc(&(t -> psi1),
         sizeof(mycomplex) * (p -> plen));
  hipMalloc(&(t -> psi2),
         sizeof(mycomplex) * (p -> plen));
  
  hipMalloc(&(t -> isJet),
         sizeof(char) * (p -> plen));

  (t -> jet_velocity)[0] = 1.0;
  (t -> jet_velocity)[1] = 0.0;
  (t -> jet_velocity)[2] = 0.0;
  
  (t -> nozzle).center[0] = 2.0 - 1.7;
  (t -> nozzle).center[1] = 1.0 - 0.034;
  (t -> nozzle).center[2] = 1.0 + 0.066;
  (t -> nozzle).len = 0.5;
  (t -> nozzle).rad = 0.5;
  

  hipMemcpyToSymbol(HIP_SYMBOL(para), t, sizeof(para_t));
}

void isf_init(Torus* p, ISF* q)
{
  p -> resx = 64;
  p -> resy = 32;
  p -> resz = 32;
  p -> sizex = 4;
  p -> sizey = 2;
  p -> sizez = 2;
  p -> plen = (p -> resx) * (p -> resy) * (p -> resz);
  
  Torus_calc_ds(p);

  q -> hbar = 0.1;
  q -> dt = 1.0 / 48.0;
  hipMalloc(&(q -> mask), 
        sizeof(mycomplex) * (p -> plen));

  hipMemcpyToSymbol(HIP_SYMBOL(torus), p, sizeof(Torus));
  hipMemcpyToSymbol(HIP_SYMBOL(isf), q, sizeof(ISF));

}



void jet_setup()
{
  Torus localtorus;
  ISF localISF;
  para_t localPara;

  isf_init(&localtorus, &localISF);
  para_init(&localtorus, &localPara);

  ISF_BuildSchroedinger<<<1,1>>>();

  set_nozzle<<<1,1>>>();



  hipDeviceSynchronize();
}
