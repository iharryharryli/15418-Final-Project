#include "hip/hip_runtime.h"
#include "ISF.cu"
#include "particle.cu"
#include "collect.cu"

struct nozzle_t
{
	double center[3];
	double len;
	double rad;
};

__constant__ nozzle_t nozzle;
nozzle_t nozzle_cpu;

__global__ void set_nozzle_and_phase_and_psi_kernel()
{
  int ind = check_limit(torus.plen);
  if(ind<0)return;

  int i,j,k;
  getCoords(ind, &i, &j, &k);
  
        double px = i * torus.dx;
        double py = j * torus.dy;
        double pz = k * torus.dz;

        double dx = px - nozzle.center[0];
        double dy = py - nozzle.center[1];
        double dz = pz - nozzle.center[2];

        if(abs(dx) <= nozzle.len / 2.0
            && (dy * dy + dz * dz) <= nozzle.rad * nozzle.rad)
        {
          para.isJet[ind] = 1;
          //printf("%d %d %d \n", i, j ,k);
        }
        else
        {
          para.isJet[ind] = 0;
        }

        para.phase[ind] = 
          para.kvec[0]*px + para.kvec[1]*py + para.kvec[2]*pz;

        para.psi1[ind] = make_hipDoubleComplex(1.0, 0.0);
        para.psi2[ind] = make_hipDoubleComplex(0.01, 0.0);


}

void set_nozzle_and_phase_and_psi()
{
  int nb = calc_numblock(torus_cpu.plen, THREADS_PER_BLOCK);
  set_nozzle_and_phase_and_psi_kernel<<<nb,THREADS_PER_BLOCK>>>();
  hipDeviceSynchronize();
}

void para_init(Torus* p, ISF* q, para_t* t, nozzle_t* n)
{
	hipMalloc(&(t -> psi1),
				 sizeof(hipDoubleComplex) * (p -> plen));
	hipMalloc(&(t -> psi2),
				 sizeof(hipDoubleComplex) * (p -> plen));

	hipMalloc(&(t -> isJet),
				 sizeof(char) * (p -> plen));

	hipMalloc(&(t -> phase),
				 sizeof(double) * (p -> plen));

	(t -> jet_velocity)[0] = 1.0;
	(t -> jet_velocity)[1] = 0.0;
	(t -> jet_velocity)[2] = 0.0;

	n->center[0] = 2.0 - 1.7;
	n->center[1] = 1.0 - 0.034;
	n->center[2] = 1.0 + 0.066;
	n->len = 0.5;
	n->rad = 0.5;

	(t -> kvec)[0] = (t -> jet_velocity)[0] / (q -> hbar);
	(t -> kvec)[1] = (t -> jet_velocity)[1] / (q -> hbar);
	(t -> kvec)[2] = (t -> jet_velocity)[2] / (q -> hbar);

	t -> omega = 0.0;
	for(int i=0; i<3; i++)
	{
		t -> omega += ((t -> jet_velocity)[i])*((t -> jet_velocity)[i]);
	}

	t -> omega /= 2.0 * (q -> hbar);


	hipMemcpyToSymbol(HIP_SYMBOL(para), t, sizeof(para_t));
	hipMemcpyToSymbol(HIP_SYMBOL(nozzle), n, sizeof(nozzle_t));
}

void isf_init(Torus* p, ISF* q)
// Includes a bunch of hard-coded values
{
  p -> resx = 64;
  p -> resy = 32;
  p -> resz = 32;
  p -> sizex = 4;
  p -> sizey = 2;
  p -> sizez = 2;
  p -> plen = (p -> resx) * (p -> resy) * (p -> resz);
  p -> yzlen = (p -> resy) * (p -> resz);
  Torus_calc_ds(p);
  hipMalloc(&(p -> div), sizeof(double) * (p -> plen));
  hipMalloc(&(p -> fftbuf), sizeof(hipDoubleComplex) * (p -> plen));
  hipfftPlan3d(&(p -> fftplan), 
      torus_cpu.resx, torus_cpu.resy, torus_cpu.resz, HIPFFT_Z2Z);


  q -> hbar = 0.1;
  q -> dt = 1.0 / 48.0;
  hipMalloc(&(q -> mask), 
        sizeof(hipDoubleComplex) * (p -> plen));


  hipMalloc(&(p -> vx),
        sizeof(double) * (p -> plen));
  hipMalloc(&(p -> vy),
        sizeof(double) * (p -> plen));
  hipMalloc(&(p -> vz),
        sizeof(double) * (p -> plen));
  
  
  hipMemcpyToSymbol(HIP_SYMBOL(torus), p, sizeof(Torus));
  hipMemcpyToSymbol(HIP_SYMBOL(isf), q, sizeof(ISF));

}

__global__ void constrain_velocity_iter(double t)
// A special procedure we need to do in order for the jet dynamics to work
{

    int ind = check_limit(torus.plen);
    if(ind < 0) return;
    /*for(int i=0; i<torus.resx; i++)
    {
      for(int j=0; j<torus.resy; j++)
      {
        for(int k=0; k<torus.resz; k++)
        {
          int ind = index3d(i,j,k);*/
          
          if(para.isJet[ind] == 1)
          {
            double amp1 = hipCabs(para.psi1[ind]);
            double amp2 = hipCabs(para.psi2[ind]);
            
            para.psi1[ind] = exp_mycomplex( 
                     make_hipDoubleComplex(0.0, para.phase[ind] - para.omega * t));
            mul_mycomplex(&para.psi1[ind], amp1);

            para.psi2[ind] = exp_mycomplex( 
                     make_hipDoubleComplex(0.0, para.phase[ind] - para.omega * t));
            mul_mycomplex(&para.psi2[ind], amp2);

          }
/*
        }
      }
    }*/
}

__global__ void print_psi()
{
		for(int i=0; i<torus.resx; i++)
		{
			for(int j=0; j<torus.resy; j++)
			{
				for(int k=0; k<torus.resz; k++)
				{
					int ind = index3d(i,j,k);
					printf("%f %f\n", para.psi1[ind].x, para.psi1[ind].y);
				}
			}
		}
}

__global__ void print_particles()
{
	for(int i=0; i<particles.num_particles; i++)
	{
		// px[i] = particles.x[i];
		// py[i] = particles.y[i];
		// pz[i] = particles.z[i];
		// printf("%f %f %f\n", particles.x[i],
		// 					particles.y[i], particles.z[i]);
	}
}

void constrain_velocity(double t)
{

    tpstart(5);
    int nb = calc_numblock(torus_cpu.plen, THREADS_PER_BLOCK); 
    constrain_velocity_iter<<<nb,THREADS_PER_BLOCK>>>(t);
    hipDeviceSynchronize();
    tpend(5);
    ISF_PressureProject();
}


__global__ void
particle_birth_kernel()
{
	for(int i=0; i<particles.num_particles; i++)
	{
		double rt = ((double)i) / particles.num_particles;
		rt *= 2 * M_PI;

		particles.x[i] = nozzle.center[0];
		particles.y[i] = nozzle.center[1] + 0.9 * nozzle.rad * cos(rt);
		particles.z[i] = nozzle.center[2] + 0.9 * nozzle.rad * sin(rt);
	}
}

void particle_birth(int num)
{

  particles_cpu.num_particles = num;
  hipMalloc(&(particles_cpu.x), sizeof(double) * num);
  hipMalloc(&(particles_cpu.y), sizeof(double) * num);
  hipMalloc(&(particles_cpu.z), sizeof(double) * num);
  
  hipMalloc(&(particles_cpu.pixel_index), sizeof(int) * num);

  hipMemcpyToSymbol(HIP_SYMBOL(particles), &particles_cpu, sizeof(particles_t)); 
  
  particle_birth_kernel<<<1,1>>>();
  hipDeviceSynchronize();
}


void jet_setup(int particleCount)
{
  // init timer
  tpinit();

  // Basic setup

  isf_init(&torus_cpu, &isf_cpu);
  para_init(&torus_cpu, &isf_cpu, &para_cpu, &nozzle_cpu);

  ISF_BuildSchroedinger();

  // Jet-specific setup
  
 
  set_nozzle_and_phase_and_psi();

  hipDeviceSynchronize();

  for(int i=0; i<10; i++)
  {
    constrain_velocity(0.0);
  }

  printf("Initialization Done! \n");

  //print_psi<<<1,1>>>();
  //hipDeviceSynchronize(); 

  // generate particles
  particle_birth(particleCount);

  // Main algorithm
  for (int i=0; i<500; i++)
  {
    // Simulate Incompressible Schroedinger Flow
    ISF_SchroedingerFlow();
    ISF_Normalize();
    ISF_PressureProject();

    constrain_velocity((i+1) * isf_cpu.dt);

    // Do particle advection

    ISF_VelocityOneForm(isf_cpu.hbar);
    Torus_StaggeredSharp();
    StaggeredAdvect();

    //printf("Iteration %d done!\n", i);
  }


  tpsummary();

  //print_psi<<<1,1>>>(); hipDeviceSynchronize();  
  //print_particles<<<1,1>>>(); hipDeviceSynchronize();
  
}
