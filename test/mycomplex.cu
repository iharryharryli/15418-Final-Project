#include "hip/hip_runtime.h"



__device__ hipDoubleComplex exp_mycomplex(hipDoubleComplex inp)
{
  hipDoubleComplex res;
  res.x = exp(inp.x) * cos(inp.y);
  res.y = exp(inp.x) * sin(inp.y);
  return res;
}

__device__ void div_mycomplex(hipDoubleComplex* n, double d)
{
  n -> x /= d;
  n -> y /= d;
}

__device__ void mul_mycomplex(hipDoubleComplex* n, double d)
{
  n -> x *= d;
  n -> y *= d;
}

__device__ double angle_mycomplex(hipDoubleComplex inp)
{
  double res =  atan2(inp.y, inp.x);
  return res;
}


