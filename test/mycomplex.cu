#include "hip/hip_runtime.h"



__device__ hipFloatComplex exp_mycomplex(hipFloatComplex inp)
{
  hipFloatComplex res;
  res.x = exp(inp.x) * cos(inp.y);
  res.y = exp(inp.x) * sin(inp.y);
  return res;
}

__device__ void div_mycomplex(hipFloatComplex* n, float d)
{
  n -> x /= d;
  n -> y /= d;
}

__device__ void mul_mycomplex(hipFloatComplex* n, float d)
{
  n -> x *= d;
  n -> y *= d;
}

__device__ float angle_mycomplex(hipFloatComplex inp)
{
  float res =  atan2(inp.y, inp.x);
  return res;
}


