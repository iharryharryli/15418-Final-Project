#include "hip/hip_runtime.h"
#include "depend.h"
#include "mycomplex.cu"
struct Torus
{
  int resx,resy,resz;
  int sizex,sizey,sizez;
  double dx,dy,dz;

  int plen;

  double* out;
  hipDoubleComplex* fftbuf;
};

void Torus_calc_ds(Torus* t)
{
  t -> dx = ((double)t -> sizex) / (t -> resx);
  t -> dy = ((double)t -> sizey) / (t -> resy);
  t -> dz = ((double)t -> sizez) / (t -> resz);
}

__constant__ Torus torus;
Torus torus_cpu;

__device__  __inline__  int 
index3d(int i, int j, int k)
{
  return (k + j*torus.resz + i*torus.resz*torus.resy);
}

__global__ void Torus_Div (double* vx, double* vy, double* vz)
{

  double dx2 = torus.dx * torus.dx;
  double dy2 = torus.dy * torus.dy;
  double dz2 = torus.dz * torus.dz;

  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ixm = (i - 1) % torus.resx;
        int iym = (j - 1) % torus.resy;
        int izm = (k - 1) % torus.resz;

        int normal_index = index3d(i,j,k);
        
        torus.out[normal_index] = 
          (vx[normal_index] - vx[index3d(ixm,j,k)])/dx2;
        torus.out[normal_index] +=
          (vy[normal_index] - vy[index3d(i,iym,k)])/dy2;
        torus.out[normal_index] +=
          (vz[normal_index] - vz[index3d(i,j,izm)])/dz2;

      }
    }
  }

}

__global__ void Torus_printfft()
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);
        printf("%f %f\n", torus.fftbuf[ind].x / torus.plen, 
              torus.fftbuf[ind].y / torus.plen);
      }
    }
  }

}

__global__ void Torus_printdouble(double* f)
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);
        printf("%f\n", f[ind]);
      }
    }
  }
}

__global__ void Torus_f2buf(double* f)
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);
        torus.fftbuf[ind] = make_hipDoubleComplex(f[ind],0.0);
       }
    }
  }
}

__global__ void PoissonSolve_main()
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);   
        double sx = sin(M_PI*i/torus.resx) / torus.dx;
        double sy = sin(M_PI*j/torus.resy) / torus.dy;  
        double sz = sin(M_PI*k/torus.resz) / torus.dz;
        double denom = sx * sx + sy * sy + sz * sz;
        double fac = 0.0;
        if(denom > 1e-16)
        {
          fac = -0.25 / denom;
        }
        //mul_mycomplex(&torus.fftbuf[ind], fac);
        torus.fftbuf[ind].x *= fac;
        torus.fftbuf[ind].y *= fac;
      }
    }
  }    
        
}

void Torus_PoissonSolve(double* f)
{
  Torus_f2buf<<<1,1>>>(f);
  hipDeviceSynchronize(); 
 

  //Torus_printfft<<<1,1>>>(); hipDeviceSynchronize(); 


  // fft
  hipfftHandle plan;
  hipfftPlan3d(&plan, torus_cpu.resx, 
              torus_cpu.resy, torus_cpu.resz, HIPFFT_Z2Z);
  hipfftExecZ2Z(plan, torus_cpu.fftbuf, 
                       torus_cpu.fftbuf, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  
  PoissonSolve_main<<<1,1>>>();
  hipDeviceSynchronize();   

  // ifft
  hipfftExecZ2Z(plan, torus_cpu.fftbuf, 
                       torus_cpu.fftbuf, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  hipfftDestroy(plan);
  

  //Torus_printfft<<<1,1>>>();
  //Torus_printdouble<<<1,1>>>(f);
  //hipDeviceSynchronize();
  

}

