#include "hip/hip_runtime.h"
#include "depend.h"
#include "mycomplex.cu"
#include "parallel.cu"
struct Torus
{
  int resx,resy,resz;
  int sizex,sizey,sizez;
  double dx,dy,dz;

  double* vx;
  double* vy;
  double* vz;

  int plen;
  int yzlen;

  double* div;
  hipDoubleComplex* fftbuf;
  hipfftHandle fftplan;

};

void Torus_calc_ds(Torus* t)
{
  t -> dx = ((double)t -> sizex) / (t -> resx);
  t -> dy = ((double)t -> sizey) / (t -> resy);
  t -> dz = ((double)t -> sizez) / (t -> resz);
}

__constant__ Torus torus;
Torus torus_cpu;

__device__  __inline__  int 
index3d(int i, int j, int k)
{
  return (k + j*torus.resz + i*torus.yzlen);
}

__device__  __inline__  void 
getCoords(int i, int *x, int *y, int *z)
{
  *x = i / (torus.yzlen);
  int t = i % torus.yzlen;
  *y = t / torus.resz;
  *z = t % torus.resz;
}

__global__ void Torus_Div ()
{
  int normal_index = check_limit(torus.plen);
  if(normal_index < 0) return;

  double dx2 = torus.dx * torus.dx;
  double dy2 = torus.dy * torus.dy;
  double dz2 = torus.dz * torus.dz;

  double* vx = torus.vx;
  double* vy = torus.vy;
  double* vz = torus.vz;

  int i,j,k;
  getCoords(normal_index, &i, &j, &k);

  /*for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {*/
        int ixm = (i - 1 + torus.resx) % torus.resx;
        int iym = (j - 1 + torus.resy) % torus.resy;
        int izm = (k - 1 + torus.resz) % torus.resz;

        
        torus.div[normal_index] = 
          (vx[normal_index] - vx[index3d(ixm,j,k)])/dx2;
        torus.div[normal_index] +=
          (vy[normal_index] - vy[index3d(i,iym,k)])/dy2;
        torus.div[normal_index] +=
          (vz[normal_index] - vz[index3d(i,j,izm)])/dz2;

      /*}
    }
  }*/

}

__global__ void Torus_printfft()
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);
        printf("%f %f\n", torus.fftbuf[ind].x / torus.plen, 
              torus.fftbuf[ind].y / torus.plen);
      }
    }
  }

}

__global__ void Torus_printdouble(double* f)
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);
        printf("%f\n", f[ind]);
      }
    }
  }
}

__global__ void Torus_div2buf()
{
  int ind = check_limit(torus.plen);
  if(ind<0)return;
          torus.fftbuf[ind] = make_hipDoubleComplex(torus.div[ind],0.0);
}

__global__ void PoissonSolve_main()
{
  int ind = check_limit(torus.plen);
  if(ind<0)return;
  int i,j,k;
  getCoords(ind,&i,&j,&k);
          double sx = sin(M_PI*i/torus.resx) / torus.dx;
        double sy = sin(M_PI*j/torus.resy) / torus.dy;  
        double sz = sin(M_PI*k/torus.resz) / torus.dz;
        double denom = sx * sx + sy * sy + sz * sz;
        double fac = 0.0;
        if(ind > 0)
        {
          fac = -0.25 / denom;
        }
        mul_mycomplex(&torus.fftbuf[ind], fac);
          
        
}

//*********** not tested! ***********
__global__ void fftshift(hipfftDoubleComplex *data)
// The thing is: fftshift for even and odd dimensional arrays 
// are really different -- the even case is much simpler than the odd case
// To save ourselves the trouble we will only implement the even fftshift
// and give an error when the input has odd dimension
{
  int xs = torus.resx / 2;
  int ys = torus.resy / 2;
  int zs = torus.resz / 2;
  int len = torus.plen;
  int x, y, z = 0;
  int j;

  /*if (len % 2 == 1){
    printf("Error: fftshift only supports even sized grid!\n");
    return;
  }*/

  int i = check_limit(len / 2);
  if(i<0) return;

  //for (int i=0; i<len/2; i++)
  //{
    hipfftDoubleComplex temp = data[i];
    getCoords(i, &x, &y, &z);
    x = (x + xs) % torus.resx;
    y = (y + ys) % torus.resy;
    z = (z + zs) % torus.resz;
    j = index3d(x, y, z);
    data[i] = data[j];
    data[j] = temp;
  //}
}

//*********** not tested! ***********
__global__ void ifftshift(hipfftDoubleComplex *data)
// Since we are only working with even-sized arrays
// ifftshift is equivalent with fftshift
{
  int xs = torus.resx / 2;
  int ys = torus.resy / 2;
  int zs = torus.resz / 2;
  int len = torus.resx * torus.resy * torus.resz;
  int x, y, z = 0;
  int j;

  if (len % 2 == 1){
    printf("Error: fftshift only supports even sized grid!\n");
    return;
  }

  for (int i=0; i<len/2; i++)
  {
    hipfftDoubleComplex temp = data[i];
    getCoords(i, &x, &y, &z);
    x = (x + xs) % torus.resx;
    y = (y + ys) % torus.resy;
    z = (z + zs) % torus.resz;
    j = index3d(x, y, z);
    data[i] = data[j];
    data[j] = temp;
  }
}

void fftn(hipfftDoubleComplex *data)
// Returns the cufft plan created
{
  tpstart(6);
  hipfftExecZ2Z(torus_cpu.fftplan, data, data, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  tpend(6);
}

void ifftn(hipfftDoubleComplex *data)
// Destorys the cufft plan after finshing
{
  tpstart(7);
  hipfftExecZ2Z(torus_cpu.fftplan, data, data, HIPFFT_BACKWARD); 
  hipDeviceSynchronize();
  tpend(7);
}

void Torus_PoissonSolve()
{
  int nb = calc_numblock(torus_cpu.plen, THREADS_PER_BLOCK);

  tpstart(8);
  Torus_div2buf<<<nb,THREADS_PER_BLOCK>>>();
  hipDeviceSynchronize(); 
  tpend(8);
 

  //Torus_printfft<<<1,1>>>(); hipDeviceSynchronize(); 

  // fft
  
  fftn(torus_cpu.fftbuf);

  // Do work in the fourier space

  tpstart(1);
  PoissonSolve_main<<<nb,THREADS_PER_BLOCK>>>();
  hipDeviceSynchronize();   
  tpend(1);

  // ifft

  ifftn(torus_cpu.fftbuf);
  

  //Torus_printfft<<<1,1>>>();
  //Torus_printdouble<<<1,1>>>(f);
}

__global__ void StaggeredSharp_kernel()
{
  int i = check_limit(torus.plen);
  if(i<0)return;
      torus.vx[i] /= torus.dx;
    torus.vy[i] /= torus.dy;
    torus.vz[i] /= torus.dz;

}

void Torus_StaggeredSharp()
{
  tpstart(9);
  int nb = calc_numblock(torus_cpu.plen, THREADS_PER_BLOCK);
  StaggeredSharp_kernel<<<nb,THREADS_PER_BLOCK>>>();
  hipDeviceSynchronize();
  tpend(9);
}


