#include "hip/hip_runtime.h"
#include "depend.h"
#include "mycomplex.cu"
struct Torus
{
  int resx,resy,resz;
  int sizex,sizey,sizez;
  double dx,dy,dz;

  double* vx;
  double* vy;
  double* vz;

  int plen;

  double* div;
  hipDoubleComplex* fftbuf;
};

void Torus_calc_ds(Torus* t)
{
  t -> dx = ((double)t -> sizex) / (t -> resx);
  t -> dy = ((double)t -> sizey) / (t -> resy);
  t -> dz = ((double)t -> sizez) / (t -> resz);
}

__constant__ Torus torus;
Torus torus_cpu;

__device__  __inline__  int 
index3d(int i, int j, int k)
{
  return (k + j*torus.resz + i*torus.resz*torus.resy);
}

__device__  __inline__  void 
getCoords(int i, int *x, int *y, int *z)
{
  *x = i / (torus.resz * torus.resy);
  *y = (i % (torus.resz * torus.resy)) / torus.resz;
  *z = (i % (torus.resz * torus.resy)) % torus.resz;
}

__global__ void Torus_Div ()
{

  double dx2 = torus.dx * torus.dx;
  double dy2 = torus.dy * torus.dy;
  double dz2 = torus.dz * torus.dz;

  double* vx = torus.vx;
  double* vy = torus.vy;
  double* vz = torus.vz;

  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ixm = (i - 1 + torus.resx) % torus.resx;
        int iym = (j - 1 + torus.resy) % torus.resy;
        int izm = (k - 1 + torus.resz) % torus.resz;

        int normal_index = index3d(i,j,k);
        
        torus.div[normal_index] = 
          (vx[normal_index] - vx[index3d(ixm,j,k)])/dx2;
        torus.div[normal_index] +=
          (vy[normal_index] - vy[index3d(i,iym,k)])/dy2;
        torus.div[normal_index] +=
          (vz[normal_index] - vz[index3d(i,j,izm)])/dz2;

      }
    }
  }

}

__global__ void Torus_printfft()
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);
        printf("%f %f\n", torus.fftbuf[ind].x / torus.plen, 
              torus.fftbuf[ind].y / torus.plen);
      }
    }
  }

}

__global__ void Torus_printdouble(double* f)
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);
        printf("%f\n", f[ind]);
      }
    }
  }
}

__global__ void Torus_div2buf()
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);
        torus.fftbuf[ind] = make_hipDoubleComplex(torus.div[ind],0.0);
       }
    }
  }
}

__global__ void PoissonSolve_main()
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ind = index3d(i,j,k);   
        double sx = sin(M_PI*i/torus.resx) / torus.dx;
        double sy = sin(M_PI*j/torus.resy) / torus.dy;  
        double sz = sin(M_PI*k/torus.resz) / torus.dz;
        double denom = sx * sx + sy * sy + sz * sz;
        double fac = 0.0;
        if(ind > 0)
        {
          fac = -0.25 / denom;
        }
        //mul_mycomplex(&torus.fftbuf[ind], fac);
        torus.fftbuf[ind].x *= fac;
        torus.fftbuf[ind].y *= fac;
      }
    }
  }    
        
}

//*********** not tested! ***********
__global__ void fftshift(hipfftDoubleComplex *data)
// The thing is: fftshift for even and odd dimensional arrays 
// are really different -- the even case is much simpler than the odd case
// To save ourselves the trouble we will only implement the even fftshift
// and give an error when the input has odd dimension
{
  int xs = torus.resx / 2;
  int ys = torus.resy / 2;
  int zs = torus.resz / 2;
  int len = torus.resx * torus.resy * torus.resz;
  int x, y, z = 0;
  int j;

  if (len % 2 == 1){
    printf("Error: fftshift only supports even sized grid!\n");
    return;
  }

  for (int i=0; i<len/2; i++)
  {
    hipfftDoubleComplex temp = data[i];
    getCoords(i, &x, &y, &z);
    x = (x + xs) % torus.resx;
    y = (y + ys) % torus.resy;
    z = (z + zs) % torus.resz;
    j = index3d(x, y, z);
    data[i] = data[j];
    data[j] = temp;
  }
}

//*********** not tested! ***********
__global__ void ifftshift(hipfftDoubleComplex *data)
// Since we are only working with even-sized arrays
// ifftshift is equivalent with fftshift
{
  int xs = torus.resx / 2;
  int ys = torus.resy / 2;
  int zs = torus.resz / 2;
  int len = torus.resx * torus.resy * torus.resz;
  int x, y, z = 0;
  int j;

  if (len % 2 == 1){
    printf("Error: fftshift only supports even sized grid!\n");
    return;
  }

  for (int i=0; i<len/2; i++)
  {
    hipfftDoubleComplex temp = data[i];
    getCoords(i, &x, &y, &z);
    x = (x + xs) % torus.resx;
    y = (y + ys) % torus.resy;
    z = (z + zs) % torus.resz;
    j = index3d(x, y, z);
    data[i] = data[j];
    data[j] = temp;
  }
}

hipfftHandle fftn(hipfftDoubleComplex *data)
// Returns the cufft plan created
{
  hipfftHandle plan;
  hipfftPlan3d(&plan, torus_cpu.resx, torus_cpu.resy, torus_cpu.resz, HIPFFT_Z2Z);
  hipfftExecZ2Z(plan, data, data, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  return plan;
}

void ifftn(hipfftDoubleComplex *data, hipfftHandle plan)
// Destorys the cufft plan after finshing
{
  hipfftExecZ2Z(plan, data, data, HIPFFT_BACKWARD); 
  hipDeviceSynchronize();
  hipfftDestroy(plan);
}

void Torus_PoissonSolve()
// TODO: This is a crazy amount of passing data back and forth...
{
  Torus_div2buf<<<1,1>>>();
  hipDeviceSynchronize(); 
 

  //Torus_printfft<<<1,1>>>(); hipDeviceSynchronize(); 

  // fft

  hipfftHandle plan = fftn(torus_cpu.fftbuf);

  // Do work in the fourier space
  PoissonSolve_main<<<1,1>>>();
  hipDeviceSynchronize();   

  // ifft

  ifftn(torus_cpu.fftbuf, plan);
  

  //Torus_printfft<<<1,1>>>();
  //Torus_printdouble<<<1,1>>>(f);
}

__global__ void StaggeredSharp_kernel()
{
  for(int i=0; i<torus.plen; i++)
  {
    torus.vx[i] /= torus.dx;
    torus.vy[i] /= torus.dy;
    torus.vz[i] /= torus.dz;

    //printf("%f\n", torus.vx[i]);
  }
}

void Torus_StaggeredSharp()
{
  StaggeredSharp_kernel<<<1,1>>>();
  hipDeviceSynchronize();
}


