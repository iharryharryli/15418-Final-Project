#include "hip/hip_runtime.h"
#include "Torus.cu"



struct ISF
{
  float hbar;
  float dt;
  hipFloatComplex* mask;
  
  float* vx;
  float* vy;
  float* vz;
};

__constant__ ISF isf;

__device__ hipFloatComplex exp_mycomplex(hipFloatComplex inp)
{
  hipFloatComplex res;
  res.x = exp(inp.x) * cos(inp.y);
  res.y = exp(inp.x) * sin(inp.y);
  return res;
}

__device__ void div_mycomplex(hipFloatComplex* n, float d)
{
  n -> x /= d;
  n -> y /= d;
}

__device__ float angle_mycomplex(hipFloatComplex inp)
{
  return atan2(inp.y, inp.x);
}

__global__ void ISF_Normalize(hipFloatComplex* psi1, hipFloatComplex* psi2)
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        float psi_norm = 
          sqrt(psi1->x*psi1->x+psi1->y*psi1->y+
               psi2->x*psi2->x+psi2->y*psi2->y);
        
        int ind = index3d(i,j,k);
        div_mycomplex(&psi1[ind], psi_norm);
        div_mycomplex(&psi2[ind], psi_norm);
      }
    }
  }
}

__global__ void ISF_BuildSchroedinger()
{
  float nx = torus.resx, ny = torus.resy, nz = torus.resz;
  float fac = -4.0 * M_PI * M_PI * isf.hbar;
  
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        float kx = (i - nx / 2) / torus.sizex;
        float ky = (j - ny / 2) / torus.sizey;
        float kz = (k - nz / 2) / torus.sizez;
        float lambda = fac * (kx * kx + ky * ky + kz * kz);
        
        int ind = index3d(i,j,k);
        
        hipFloatComplex inp;
        inp.x = 0;
        inp.y = lambda * isf.dt / 2;
        
        isf.mask[index3d(i,j,k)] = exp_mycomplex(inp);

        //printf("%f %f \n", isf.mask[index3d(i,j,k)].x,isf.mask[index3d(i,j,k)].y);
        
      }
    }
  }

  printf("Done ISF_BuildSchroedinger \n"); 
}

__global__ void ISF_VelocityOneForm(hipFloatComplex* psi1, 
                                    hipFloatComplex* psi2, 
                                  float hbar)
{
  for(int i=0; i<torus.resx; i++)
  {
    for(int j=0; j<torus.resy; j++)
    {
      for(int k=0; k<torus.resz; k++)
      {
        int ixp = (i + 1) % torus.resx;
        int iyp = (j + 1) % torus.resy;
        int izp = (k + 1) % torus.resz;

        int ind = index3d(i,j,k);
        int vxi = index3d(ixp,j,k);
        int vyi = index3d(i,iyp,k);
        int vzi = index3d(i,j,izp);

        hipFloatComplex vxraw = hipCaddf(
          hipCmulf(hipConjf(psi1[ind]),psi1[ixp]),
          hipCmulf(hipConjf(psi2[ind]),psi2[ixp])
          );
        hipFloatComplex vyraw = hipCaddf(
          hipCmulf(hipConjf(psi1[ind]),psi1[iyp]),
          hipCmulf(hipConjf(psi2[ind]),psi2[iyp])
          );
        hipFloatComplex vzraw = hipCaddf(
          hipCmulf(hipConjf(psi1[ind]),psi1[izp]),
          hipCmulf(hipConjf(psi2[ind]),psi2[izp])
          );

        isf.vx[ind] = angle_mycomplex(vxraw);
        isf.vy[ind] = angle_mycomplex(vyraw);
        isf.vz[ind] = angle_mycomplex(vzraw);


      }
    }
  }
}


