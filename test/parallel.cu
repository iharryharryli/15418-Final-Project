
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256

int calc_numblock(int limit, int threadsPerBlock)
{
  return (limit + threadsPerBlock - 1) / threadsPerBlock;
}

__device__ int check_limit(int limit)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(i < limit)
    return i;
  return -1;
}
