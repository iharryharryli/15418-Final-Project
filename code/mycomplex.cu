#include "hip/hip_runtime.h"



__device__ __inline__ 
hipDoubleComplex exp_mycomplex(hipDoubleComplex inp)
{
  hipDoubleComplex res;
  res.x = exp(inp.x) * cos(inp.y);
  res.y = exp(inp.x) * sin(inp.y);
  return res;
}

__device__ __inline__ 
void div_mycomplex(hipDoubleComplex* n, double d)
{
  n -> x /= d;
  n -> y /= d;
}

__device__ __inline__
void mul_mycomplex(hipDoubleComplex* n, double d)
{
  n -> x *= d;
  n -> y *= d;
}

__device__ __inline__
double angle_mycomplex(hipDoubleComplex inp)
{
  double res =  atan2(inp.y, inp.x);
  return res;
}


